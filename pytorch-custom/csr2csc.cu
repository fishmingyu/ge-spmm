#include <hip/hip_runtime.h>
#include <torch/types.h>
#include <hipsparse.h>

#define checkCudaError( a ) do { \
    if (hipSuccess != (a)) { \
    fprintf(stderr, "Cuda runTime error in line %d of file %s \
    : %s \n", __LINE__, __FILE__, hipGetErrorString(hipGetLastError()) ); \
    exit(EXIT_FAILURE); \
    } \
} while(0)

#define checkCuSparseError( a ) do { \
    if (HIPSPARSE_STATUS_SUCCESS != (a)) { \
    fprintf(stderr, "CuSparse runTime error in line %d of file %s \
    : %s \n", __LINE__, __FILE__, hipGetErrorString(hipGetLastError()) ); \
    exit(EXIT_FAILURE); \
    } \
} while (0)

void csr2cscKernel(int m, int n, int nnz,
    int *csrRowPtr, int *csrColInd, float *csrVal,
    int *cscColPtr, int *cscRowInd, float *cscVal
)
{
    hipsparseHandle_t handle;
    size_t bufferSize = 0;
    void* buffer = NULL;
    checkCuSparseError(hipsparseCsr2cscEx2_bufferSize(handle,
        m,
        n,
        nnz,
        csrVal,
        csrRowPtr,
        csrColInd,
        cscVal,
        cscColPtr,
        cscRowInd,
        HIP_R_32F,
        HIPSPARSE_ACTION_SYMBOLIC,
        HIPSPARSE_INDEX_BASE_ZERO,
        HIPSPARSE_CSR2CSC_ALG1,
        &bufferSize
    ));
    checkCudaError(hipMalloc((void**)&buffer, bufferSize * sizeof(float)));
    checkCuSparseError(hipsparseCsr2cscEx2(handle,
        m,
        n,
        nnz,
        csrVal,
        csrRowPtr,
        csrColInd,
        cscVal,
        cscColPtr,
        cscRowInd,
        HIP_R_32F,
        HIPSPARSE_ACTION_NUMERIC,
        HIPSPARSE_INDEX_BASE_ZERO,
        HIPSPARSE_CSR2CSC_ALG1,
        buffer
    ));
    checkCudaError(hipFree(buffer));
}

torch::Tensor csr2csc_cuda(
    torch::Tensor csrRowPtr,
    torch::Tensor csrColInd,
    torch::Tensor csrVal,
    torch::Tensor cscColPtr,
    torch::Tensor cscRowInd
)
{
    const auto m = csrRowPtr.size(0) - 1;
    const auto n = cscColPtr.size(0) - 1;
    const auto nnz = csrColInd.size(0);
    auto devid = csrRowPtr.device().index();
    auto options = torch::TensorOptions().dtype(torch::kFloat32).device(torch::kCUDA, devid);
    auto cscVal = torch::empty({nnz}, options);
    csr2cscKernel(m, n, nnz, csrRowPtr.data_ptr<int>(), csrColInd.data_ptr<int>(), csrVal.data_ptr<float>(), 
    cscColPtr.data_ptr<int>(), cscRowInd.data_ptr<int>(), cscVal.data_ptr<float>());
    return cscVal;
}

